#include <chrono>
#include <iostream>
#include "../../src/kernels/add/vectorAddKernel.cuh"
#include "hipblas.h"

using namespace std::chrono;
using namespace kernel;

// refer to https://developer.download.nvidia.com/video/gputechconf/gtc/2019/presentation/s9956-best-practices-when-benchmarking-cuda-applications_V2.pdf
// to set proper environment to run benchmark
#define REGISTER_KERNEL_BENCHMARK(kernel) \
    void kernel##BenchMark() \
    {   \
        int N = 1 << 20;    \
        int iter_num = 100; \
        int nBytes = N * sizeof(float); \
        float* x_device;    \
        hipMalloc(&x_device, nBytes);  \
        hipMemset(&x_device, 0, nBytes);    \
        float* y_device;    \
        hipMalloc(&y_device, nBytes);  \
        hipMemset(&y_device, 0, nBytes);    \
        float* z_device;    \
        hipMalloc(&z_device, nBytes);  \
        hipMemset(&z_device, 0, nBytes);    \
        dim3 blockSize(256);    \
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x); \
        for (size_t i = 0; i < 10; i++) \
        {   \
            kernel<<<gridSize, blockSize>>>(x_device, y_device, z_device, N);   \
        }   \
        auto start = steady_clock::now();   \
        for (size_t i = 0; i < iter_num; i++)   \
        {   \
            kernel<<<gridSize, blockSize>>>(x_device, y_device, z_device, N);   \
        }   \
        auto end = steady_clock::now(); \
        auto usecs = duration_cast<duration<float, milliseconds::period>>(end - start); \
        std::cout << "kernel execution time: " << usecs.count() / iter_num << " ms" << std::endl;   \
        hipblasHandle_t handle = 0;  \
        float alpha = 1.0f; \
        hipblasCreate(&handle);  \
        for (size_t i = 0; i < 10; i++) \
        {   \
            hipblasSaxpy(handle, N, &alpha, x_device, 1, y_device, 1); \
        }   \
        start = steady_clock::now();   \
        for (size_t i = 0; i < iter_num; i++)   \
        {   \
            hipblasSaxpy(handle, N, &alpha, x_device, 1, y_device, 1); \
        }   \
        end = steady_clock::now(); \
        usecs = duration_cast<duration<float, milliseconds::period>>(end - start); \
        std::cout << "cublas execution time: " << usecs.count() / iter_num << " ms" << std::endl;   \
        hipFree(x_device); \
        hipFree(y_device); \
        hipFree(z_device); \
        hipblasDestroy(handle); \
    }   \

REGISTER_KERNEL_BENCHMARK(add);

int main()
{
    addBenchMark();
    return 0;
}